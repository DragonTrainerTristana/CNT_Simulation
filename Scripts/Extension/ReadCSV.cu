// 필요한 header 파일 불러오기

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>

// CUDA를 사용하기 위한 header 파일 불러오기
// #include <cuda.h>
// #include <cuda_runtime.h>



// CUDA 없이 CPU에서만 코드를 돌릴 때에도 에러 없이 돌리기 위한 부분
// CUDACC가 정의되지 않았으면 = CUDA가 사용 불가능하면
// __host__와 __device__를 빈 macro로 지정
// 코드에서 삭제되어서 에러 발생 X
#ifndef __CUDACC__
#define __host__
#define __device__
#endif


// 3차원 vector struct 정의
struct Vector3 {

    // 3차원에서 점을 나타내는 좌표 저장장
    float x, y, z;

    // 기본 생성자
    __host__ __device__ Vector3() : x(0), y(0), z(0) {}

    // 사용자 정의 생성자
    __host__ __device__ Vector3(float X, float Y, float Z) : x(X), y(Y), z(Z) {}

};


// CNT는 1개의 선으로 표현
// 선은 시작점과 끝점으로 구성됨
// .csv 파일의 구조를 고려해서 좌측에 끝점
// 우측에 시작점이 오도록 struct를 정의
struct CNT {

    Vector3 end;
    Vector3 start;

};



// 
__global__ void processFibers(CNT* fibers, int size) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < size) {
        // CUDA에서 points[idx]에 접근 가능
        Vector3 End = fibers[idx].end;
        Vector3 Start = fibers[idx].start;

        printf("Start : (%f, %f, %f), End : (%f, %f, %f)\n",
                Start.x, Start.y, Start.z, End.x, End.y, End.z);
    }
}



// filename이라는 변수와 일치하는 파일 명을 가지는 .csv 파일을 line별로 (row별로) 읽어옴
// 현재 읽고 있는 row에 저장되어 있는 정보를 CNT 구조체의 형태로 저장
// 이후 해당 CNT 구조체를 std::vector라는 container 안에 저장
// 파일 안의 모든 데이터를 읽을 때까지 이를 계속 반복하는 함수
std::vector<CNT> readCSV (const std::string& filename) 
{

    // .csv 파일의 필요한 모든 정보를 담기 위한 대상 CNT 정의
    std::vector<CNT> fibers;

    // .csv 파일 parsing을 위한 도구 불러오기
    std::ifstream file(filename);


    // 파일이 열리지 않으면
    if (!file.is_open()) {

        // 에러 출력 
        std::cerr << "Error : Unable to Open File" << filename << std::endl;

        // 함수의 return 자료형을 맞춰주기 위한 return 값 설정
        // 실제로는 이 경우 file을 읽어오지 못한 경우이므로 points는 비어 있을 것임
        return fibers;

    }


    // 파일의 정보를 한 줄씩 읽을 때 이를 저장할 대상인 line 지정
    std::string line;

    if(std::getline(file, line)) {
        // 첫 번째 줄을 읽지만 아무 것도 하지 않음으로써 무시
    }


    // 앞선 if문을 통해서
    // getline이 첫 번째 줄에 대해서는 이미 실행되었으므로
    // while 문의 getline은 두 번째 줄부터 읽기 시작함
    while(std::getline(file, line)) {

        // 각 줄(row)에 대해서 아래 내용 반복

        // .csv 파일 parsing을 위한 도구 불러오기
        // line 안에 저장된 데이터를 stream으로 불러옴
        // 이제 ss라는 대상 안에는 데이터가 담긴 line stream이 저장됨
        std::stringstream ss(line);
        // line stream에서 임의의 구분자로 구분된 데이터를 불러올 때
        // 구분자로 구분되는 최소의 단위 데이터를 저장하기 위한 대상 token 정의
        std::string token;

        // 한 줄(row)의 데이터를 저장하기 위한 대상 values 정의
        std::vector<float> values;

        // ss에 저장된 line stream에서부터 
        // 쉼표를 구분자로 하여 데이터를 불러오는 함수 std::getline(ss, token, ',')

        // 읽고 있는 줄의 첫 번째 column에는 segment 번호가 들어 있으므로 무시
        std::getline(ss, token, ',');

        // 읽고 있는 줄의 나머지 여섯 개 column을 읽은 뒤
        // 이를 values에 저장함
        while (std::getline(ss, token, ',')) {
            values.push_back(std::stof(token));
        }

        // values에 저장된 대상을 CNT 자료형을 가진 대상인 fiber로 옮기고
        // 이를 다시 .csv 파일의 전체 정보를 담는 fibers로 옮김
        if (values.size() == 6) {
            CNT fiber;
            fiber.end = Vector3(values[0], values[1], values[2]);
            fiber.start = Vector3(values[3], values[4], values[5]);
            fibers.push_back(fiber);
        }

    }


    // 파일 닫아주고
    file.close();

    // .csv 파일 안의 필요한 정보 return
    return fibers;

}



int main () 
{

    std::string filename = "FiberSegments6000-1.csv";
    std::vector<CNT> CNTs = readCSV(filename);

    #ifdef __HIPCC__

        CNT* d_CNTs;
        size_t data_size = CNTs.size() * sizeof(CNT);

        hipMalloc(&d_CNTs, data_size);
        hipMemcpy(d_CNTs, CNTs.data(), data_size, hipMemcpyHostToDevice);

        // CUDA 커널 호출 (여기서 각 점을 처리)
        int blockSize = 256;
        int numBlocks = (CNTs.size() + blockSize - 1) / blockSize;

        // 
        processFibers<<<numBlocks, blockSize>>>(d_CNTs, CNTs.size());

        // 커널 실행 후 결과 확인
        hipDeviceSynchronize();

        // GPU 메모리 해제
        hipFree(d_CNTs);

    #else

        for (const auto& fiber : CNTs) {
            std::cerr << "Number of CNTs: " << CNTs.size() << std::endl;
            std::cout << " Start : (" << fiber.start.x << ", " << fiber.start.y << ", " << fiber.start.z << ")"
                    << " End : (" << fiber.end.x << ", " << fiber.end.y << ", " << fiber.end.z << ")" << std::endl;
        }

    #endif
    
    return 0;

}
