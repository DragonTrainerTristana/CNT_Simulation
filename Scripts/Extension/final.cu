#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <cmath>

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif

#ifndef __HIPCC__
#define __host__
#define __device__
#endif

struct Vector3 {
    float x, y, z;
    __host__ __device__ Vector3() : x(0), y(0), z(0) {}
    __host__ __device__ Vector3(float X, float Y, float Z) : x(X), y(Y), z(Z) {}
    __host__ __device__ float norm() const { return sqrtf(x * x + y * y + z * z); }
    __host__ __device__ Vector3 operator+(const Vector3& other) const { return Vector3(x + other.x, y + other.y, z + other.z); }
    __host__ __device__ Vector3 operator-(const Vector3& other) const { return Vector3(x - other.x, y - other.y, z - other.z); }
    template <typename T> __host__ __device__ Vector3 operator*(const T scalar) const { return Vector3(x * scalar, y * scalar, z * scalar); }
    template <typename T> __host__ __device__ Vector3 operator/(const T scalar) const { return Vector3(x / scalar, y / scalar, z / scalar); }
    __host__ __device__ Vector3 operator+=(const Vector3& other) { x += other.x; y += other.y; z += other.z; return *this; }
    __host__ __device__ Vector3 operator-=(const Vector3& other) { x -= other.x; y -= other.y; z -= other.z; return *this; }
    template <typename T> __host__ __device__ Vector3 operator*=(const T scalar) { x *= scalar; y *= scalar; z *= scalar; return *this; }
    template <typename T> __host__ __device__ Vector3 operator/=(const T scalar) { x /= scalar; y /= scalar; z /= scalar; return *this; }
    __host__ __device__ void clamp(float minVal) {z = fmaxf(z, minVal);}
};

struct CNT {
    Vector3 start;
    Vector3 end;
};

using matrix = std::vector<std::vector<char>>;

inline char toBit(const std::string& token) { return token == "S" ? 1 : 0; }

std::vector<CNT> readCSV1(const std::string& filename) {
    std::vector<CNT> fibers;
    std::ifstream file(filename);
    if (!file.is_open()) return fibers;
    std::string line;
    std::getline(file, line);
    std::vector<CNT> temp;
    while (std::getline(file, line)) {
        if (line.empty()) continue;
        std::stringstream ss(line);
        std::string token;
        std::vector<float> values;
        std::getline(ss, token, ',');
        while (std::getline(ss, token, ',')) {
            try { values.push_back(std::stof(token)); } catch (...) { values.clear(); break; }
        }
        if (values.size() != 6) continue;
        temp.push_back({Vector3(values[0], values[1], values[2]), Vector3(values[3], values[4], values[5])});
    }
    file.close();
    for (size_t i = 0; i + 7 < temp.size(); i += 8) fibers.push_back({temp[i].start, temp[i + 7].end});
    return fibers;
}

matrix readCSV2(const std::string& filename) {
    matrix mat;
    std::ifstream file(filename);
    if (!file.is_open()) return mat;
    std::string line;
    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string token;
        std::vector<char> row;
        while (std::getline(ss, token, ',')) row.push_back((token == "S") ? 1 : token[0] - '0');
        mat.push_back(std::move(row));
    }
    return mat;
}

std::vector<char> flatten(const matrix& mat) {
    std::vector<char> flattened;
    for (const auto& row : mat) flattened.insert(flattened.end(), row.begin(), row.end());
    return flattened;
}

#ifdef __HIPCC__
__global__ void Force(CNT* d_Force, CNT* d_CNTs, char* d_flattened, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    d_Force[i] = CNT();
    if (d_flattened[i * (size + 2)] == 1) d_Force[i].start.x += -1.0f;
    if (d_flattened[i * (size + 2) + size + 1] == 1) d_Force[i].end.x += 1.0f;
    Vector3 diff = d_CNTs[i].start - d_CNTs[i].end;
    const auto L = 200;
    float difference = (diff.norm() - L) / L;
    Vector3 springforce = diff * difference;
    d_Force[i].start += springforce * -1.0f;
    d_Force[i].end += springforce;
}

__global__ void TakeStep(float dt, CNT* d_Force, CNT* d_CNTs, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    float half_dt2 = 0.5f * dt * dt;
    d_CNTs[i].start += d_Force[i].start * half_dt2;
    d_CNTs[i].end += d_Force[i].end * half_dt2;
}

__global__ void correct (CNT& d_CNTs, char* d_flattened, int size)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= size) return;

    d_CNTs.start.clamp(0.0f);
    d_CNTs.end.clamp(0.0f);

    // 수정 필요...
    for (int j = 1 + i * (size + 2); j < (i + 1) * (size + 2) - 1; ++j) {
        if (d_flattened[j] == 1){
            int idx = j - i * (size + 2) - 1;
            if (idx == i) {}
            else {
                Vector3 diff1 = d_CNTs[i].start - d_CNTs[idx].end;
                Vector3 diff2 = d_CNTs[i].end - d_CNTs[idx].start;
                if (diff1.norm() < diff2.norm()) {
                    diff1 /= 2.0f;
                    d_CNTs[i].start += diff1;
                    d_CNTs[idx].end -= diff1;
                }
                else if (diff2.norm() < diff1.norm()) {
                    diff2 /= 2.0f;
                    d_CNTs[i].start += diff2;
                    d_CNTs[idx].end -= diff2;
                }
            }
        }
    }
}
#endif


void ForceCPU(std::vector<CNT>& Forces, const std::vector<CNT>& CNTs, const std::vector<char>& flattened, int size) {
    const int cols = size + 2;
    for (int i = 0; i < size; ++i) {
        Forces[i] = CNT();
        if (i * cols >= flattened.size() || (i * cols + size + 1) >= flattened.size()) continue;
        if (flattened[i * cols] == 1) Forces[i].start.x += -1.0f;
        if (flattened[i * cols + size + 1] == 1) Forces[i].end.x += 1.0f;
        Vector3 diff = CNTs[i].start - CNTs[i].end;
        const auto L = 200;
        float difference = (diff.norm() - L) / L;

        Vector3 springforce = diff * difference;
        Forces[i].start += springforce * -1.0f;
        Forces[i].end += springforce;
    }
}

void TakeStepCPU(float dt, std::vector<CNT>& Forces, std::vector<CNT>& CNTs, int size) {
    const float mass = 1.0f;
    const float half_dt2 = 0.5f * dt * dt;
    for (int i = 0; i < size; ++i) {
        CNTs[i].start += Forces[i].start * (half_dt2 / mass);
        CNTs[i].end += Forces[i].end * (half_dt2 / mass);
    }
}

void correctCPU (std::vector<CNT>& CNTs, const std::vector<char>& flattened, int size) 
{
    for (int i = 0; i < size; ++i) {
        CNTs[i].start.clamp(0.0f);
        CNTs[i].end.clamp(0.0f);

        for (int j = 1 + i * (size + 2); j < (i + 1) * (size + 2) - 1; ++j) {
            if (flattened[j] == 1) {
                int idx = j - i * (size + 2) - 1;
                if (idx == i) {}
                else {
                    Vector3 diff1 = CNTs[i].start - CNTs[idx].end;
                    Vector3 diff2 = CNTs[i].end - CNTs[idx].start;
                    if (diff1.norm() < diff2.norm()) {
                        diff1 /= 10.0f;
                        CNTs[i].start += diff1;
                        CNTs[idx].end -= diff1;
                    }
                    else if (diff2.norm() < diff1.norm()) {
                        diff2 /= 10.0f;
                        CNTs[i].start += diff2;
                        CNTs[idx].end -= diff2;
                    }
                }
            }
        }
    }
}



void MakeCSV(const CNT* d_CNTs, int size, const std::string& filename) {
    std::vector<CNT> h_CNTs(size);
#ifdef __HIPCC__
    hipMemcpy(h_CNTs.data(), d_CNTs, sizeof(CNT) * size, hipMemcpyDeviceToHost);
#else
    for (int i = 0; i < size; ++i) h_CNTs[i] = d_CNTs[i];
#endif
    std::ofstream file(filename);
    if (!file.is_open()) return;
    file << "Fiber Segment Index, Start X, Start Y, Start Z, End X, End Y, End Z\n";
    for (int i = 0; i < size; ++i) {
        const auto& cnt = h_CNTs[i];
        file << i << "," << cnt.start.x << "," << cnt.start.y << "," << cnt.start.z << ","
             << cnt.end.x << "," << cnt.end.y << "," << cnt.end.z << "\n";
    }
    file.close();
}

int main() {
    std::string filename1 = "FiberSegments6000.csv";
    std::string filename2 = "CollisionCheck6000.csv";
    std::string output = "Result.csv";

    std::vector<CNT> CNTs = readCSV1(filename1);
    matrix mat = readCSV2(filename2);
    std::vector<char> flattened = flatten(mat);
    int size = CNTs.size();
    if (size == 0) return -1;

#ifdef __HIPCC__
    CNT* d_CNTs;
    CNT* d_Forces;
    char* d_flattened;
    hipMalloc(&d_CNTs, sizeof(CNT) * size);
    hipMalloc(&d_Forces, sizeof(CNT) * size);
    hipMalloc(&d_flattened, sizeof(char) * flattened.size());
    hipMemcpy(d_CNTs, CNTs.data(), sizeof(CNT) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_flattened, flattened.data(), sizeof(char) * flattened.size(), hipMemcpyHostToDevice);
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    const float dt = 0.01f;
    const int num_steps = 10;
    for (int step = 0; step < num_steps; ++step) {
        Force<<<blocksPerGrid, threadsPerBlock>>>(*d_Forces, *d_CNTs, *d_flattened, size);
        hipDeviceSynchronize();
        TakeStep<<<blocksPerGrid, threadsPerBlock>>>(dt, *d_Forces, *d_CNTs, size);
        hipDeviceSynchronize();
        correct<<<blocksPerGrid, threadsPerBlock>>>(*d_CNTs, d_flattened, size);
        hipDeviceSynchronize();
    }
    MakeCSV(d_CNTs, size, output);
    hipFree(d_CNTs);
    hipFree(d_Forces);
    hipFree(d_flattened);
#else
    std::vector<CNT> Forces(size);
    const float dt = 0.01f;
    const int num_steps = 10;
    for (int step = 0; step < num_steps; ++step) {
        ForceCPU(Forces, CNTs, flattened, size);
        TakeStepCPU(dt, Forces, CNTs, size);
        //correctCPU(CNTs, flattened, size);
    }
    MakeCSV(CNTs.data(), size, output);
#endif
    return 0;
}